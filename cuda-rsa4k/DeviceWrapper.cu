#include "hip/hip_runtime.h"
#include "DeviceWrapper.h"
#include "BigInteger.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

typedef struct
{
	unsigned int value;	
	unsigned int padding[31];
	// padding to match with 32 byte memory line

} memory32byte;

typedef struct
{
	memory32byte result[DeviceWrapper::ADDITION_CELLS_PER_THREAD];
	unsigned int carry;	
	// 4 byte carry offsets to another memory bank, which eliminates bank conflicts

} additionSharedMemory;

//parallel multiplication config
static const int EVEN_EVEN = 0x0;
static const int EVEN_ODD = 0x1;
static const int ODD_EVEN = 0x2;
static const int ODD_ODD = 0x3;

__host__ __device__ inline int isXodd(int config)
{
	return ((0xFFFFFFFD | config) == 0xFFFFFFFF) ? 1 : 0;
}

__host__ __device__ inline int isYodd(int config)
{
	return ((0xFFFFFFFE | config) == 0xFFFFFFFF) ? 1 : 0;
}

extern "C" __global__ void device_add(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	// implementation in DeviceWrapper.ptx	
}

extern "C" __global__ void device_multiply(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	// implementation in DeviceWrapper.ptx	
}

extern "C" __global__ void device_get_clock(unsigned long long* result)
{
	// implementation in DeviceWrapper.ptx	
}

__constant__ unsigned int deviceIndexFixupTable[129];

extern "C" __global__ void device_multiply_partial(unsigned int* result, const unsigned int* x, const unsigned int* y, const int config)
{
	register const int arraySize = BigInteger::ARRAY_SIZE + 1;
	register const int sharedMemoryLines = DeviceWrapper::MULTIPLICATION_THREAD_COUNT + 2;
	register const int memoryBanksCount = 32;

	__shared__ unsigned int sharedResult[memoryBanksCount * sharedMemoryLines];
	__shared__ unsigned int carries[memoryBanksCount * sharedMemoryLines];

	register const int xIndex = threadIdx.x * 2 + isXodd(config);

	sharedResult[deviceIndexFixupTable[xIndex]] = 0;
	sharedResult[deviceIndexFixupTable[xIndex + 1]] = 0;
	carries[deviceIndexFixupTable[xIndex]] = 0;
	carries[deviceIndexFixupTable[xIndex + 1]] = 0;

#pragma unroll
	for (register int yIndex = isYodd(config); yIndex < arraySize; yIndex = yIndex + 2)
	{
		if (xIndex + yIndex >= arraySize)
			break;

		register unsigned int carry = carries[deviceIndexFixupTable[xIndex + yIndex]];
		carries[deviceIndexFixupTable[xIndex + yIndex]] = 0;

		asm volatile (
			"add.cc.u32 %0, %0, %5; \n\t"
			"mad.lo.cc.u32 %0, %3, %4, %0; \n\t"
			"madc.hi.cc.u32 %1, %3, %4, %1; \n\t"
			"addc.u32 %2, %2, 0; \n\t"
			: "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex]]), "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex + 1]]), "+r"(carries[deviceIndexFixupTable[xIndex + yIndex + 2]])
			: "r"(x[xIndex]), "r"(y[yIndex]), "r"(carry));

		__syncthreads();
	}
			
	result[xIndex] = sharedResult[deviceIndexFixupTable[xIndex]];
	result[xIndex + 1] = sharedResult[deviceIndexFixupTable[xIndex + 1]];

	__syncthreads();
}

extern "C" __global__ void device_add_partial(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	register const int resultIndex = threadIdx.x;
	register const int startIndex = resultIndex * DeviceWrapper::ADDITION_CELLS_PER_THREAD;

	__shared__ additionSharedMemory shared[BigInteger::ARRAY_SIZE / DeviceWrapper::ADDITION_CELLS_PER_THREAD + 1];		
	
	register int index;

#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
	{
		asm volatile (
			"addc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate carry
			: "=r"(shared[resultIndex].result[index].value)
			: "r"(x[startIndex + index]), "r"(y[startIndex + index]));		
	}

	// last iteration generates and stores carry in the array
	asm volatile (
		"addc.cc.u32 %0, %2, %3; \n\t"
		"addc.u32 %1, 0, 0; \n\t"
		: "=r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]));

	__syncthreads();	

	register unsigned int carry;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ADDITION_THREAD_COUNT; i++)
	{
		index = 0;
		carry = shared[resultIndex].carry;

		// first iteration propagates carry from array
		asm volatile (
			"add.cc.u32 %0, %0, %1; \n\t"	//  
			: "+r"(shared[resultIndex].result[index].value)
			: "r"(carry));

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"addc.cc.u32 %0, %0, 0; \n\t"	//propagate generated carries
				: "+r"(shared[resultIndex].result[index].value));
		}

		// last iteration generates and stores carry in the array
		asm volatile (
			"addc.cc.u32 %0, %0, 0; \n\t"
			"addc.u32 %1, 0, 0; \n\t"
			: "+r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry));

		__syncthreads();
	}	

#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD; index++)
	{
		result[startIndex + index] = shared[resultIndex].result[index].value;
	}

	__syncthreads();
}

inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	}
	return result;
}

DeviceWrapper::DeviceWrapper()
{
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(deviceIndexFixupTable), indexFixupTable, sizeof(unsigned int) * 129));
}

DeviceWrapper::~DeviceWrapper()
{
	delete[] indexFixupTable;
}

unsigned long long DeviceWrapper::getClock(void)
{
	unsigned long long clock;
	unsigned long long* deviceClock;
	checkCuda(hipMalloc(&deviceClock, sizeof(unsigned long long)));
	
	device_get_clock << <1, 1 >> > (deviceClock);

	checkCuda(hipMemcpy(&clock, deviceClock, sizeof(unsigned long long), hipMemcpyDeviceToHost));
	checkCuda(hipFree(deviceClock));
	
	return clock;
}

unsigned int* DeviceWrapper::add(const BigInteger& x, const BigInteger& y)
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];	

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;
	
	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size + sizeof(unsigned int)));	// + 1 to check for overflow
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));
	
	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	
	device_add << <1, 1 >> > (device_result, device_x, device_y);
	
	checkCuda(hipMemcpy(resultArray, device_result, size, hipMemcpyDeviceToHost));

	unsigned int overflow;
	checkCuda(hipMemcpy(&overflow, device_result + BigInteger::ARRAY_SIZE, sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::add overflow!" << endl;
		throw std::overflow_error("BigInteger::add overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));
	
	return resultArray;
}

unsigned int* DeviceWrapper::addParallel(const BigInteger& x, const BigInteger& y)
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];	

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size + sizeof(unsigned int))); // + 1 to check for overflow
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT >> > (device_result, device_x, device_y);

	checkCuda(hipMemcpy(resultArray, device_result, size, hipMemcpyDeviceToHost));

	unsigned int overflow;
	checkCuda(hipMemcpy(&overflow, device_result + BigInteger::ARRAY_SIZE, sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::add overflow!" << endl;
		throw std::overflow_error("BigInteger::add overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	return resultArray;
}

unsigned int* DeviceWrapper::multiply(const BigInteger& x, const BigInteger& y)
{	
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];
	
	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size + sizeof(unsigned int)));	// + 1 to check for overflow
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));
		
	device_multiply << <1, 1 >> > (device_result, device_x, device_y);

	checkCuda(hipMemcpy(resultArray, device_result, size, hipMemcpyDeviceToHost));
	
	unsigned int overflow;
	checkCuda(hipMemcpy(&overflow, device_result + BigInteger::ARRAY_SIZE, sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		throw std::overflow_error("BigInteger::multiply overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	return resultArray;
}

unsigned int* DeviceWrapper::multiplyParallel(const BigInteger& x, const BigInteger& y)
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;
	int deviceResultArraySize = size + sizeof(unsigned int);

	unsigned int* device_result_even_even;
	unsigned int* device_result_even_odd;
	unsigned int* device_result_odd_even;
	unsigned int* device_result_odd_odd;

	unsigned int* device_result_even;
	unsigned int* device_result_odd;

	unsigned int* device_result;

	unsigned int* device_x;
	unsigned int* device_y;


	// device memory allocations
	checkCuda(hipMalloc(&device_result_even_even, deviceResultArraySize));
	checkCuda(hipMalloc(&device_result_even_odd, deviceResultArraySize));
	checkCuda(hipMalloc(&device_result_odd_even, deviceResultArraySize));
	checkCuda(hipMalloc(&device_result_odd_odd, deviceResultArraySize));

	checkCuda(hipMalloc(&device_result_even, deviceResultArraySize));
	checkCuda(hipMalloc(&device_result_odd, deviceResultArraySize));
	
	checkCuda(hipMalloc(&device_result, deviceResultArraySize));

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	// two parallel streams for indepentent computations
	hipStream_t evenStream, oddStream;
	checkCuda(hipStreamCreate(&evenStream));
	checkCuda(hipStreamCreate(&oddStream));

	// kernel launches
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, evenStream >> > (device_result_even_even, device_x, device_y, EVEN_EVEN);
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, evenStream >> > (device_result_even_odd, device_x, device_y, EVEN_ODD);
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, oddStream >> > (device_result_odd_even, device_x, device_y, ODD_EVEN);
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, oddStream >> > (device_result_odd_odd, device_x, device_y, ODD_ODD);

	// reduction
	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT, 0, evenStream >> > (device_result_even, device_result_even_even, device_result_even_odd);
	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT, 0, oddStream >> > (device_result_odd, device_result_odd_even, device_result_odd_odd);

	checkCuda(hipStreamSynchronize(evenStream));
	checkCuda(hipStreamSynchronize(oddStream));

	// reduction
	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT >> > (device_result, device_result_even, device_result_odd);
	
	// copy result to the host
	checkCuda(hipMemcpy(resultArray, device_result, size, hipMemcpyDeviceToHost));

	// kill streams
	checkCuda(hipStreamDestroy(evenStream));
	checkCuda(hipStreamDestroy(oddStream));
	
	unsigned int overflow;
	checkCuda(hipMemcpy(&overflow, device_result + BigInteger::ARRAY_SIZE, sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		//throw std::overflow_error("BigInteger::multiply overflow");
	}

	// clear memory
	checkCuda(hipFree(device_result_even_even));
	checkCuda(hipFree(device_result_even_odd));
	checkCuda(hipFree(device_result_odd_even));
	checkCuda(hipFree(device_result_odd_odd));

	checkCuda(hipFree(device_result_even));
	checkCuda(hipFree(device_result_odd));

	checkCuda(hipFree(device_result));

	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	// todo overflow?

	return resultArray;
}


