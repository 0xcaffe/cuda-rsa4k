#include "hip/hip_runtime.h"
#include "DeviceWrapper.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

//parallel multiplication config
static const int EVEN_EVEN = 0x0;
static const int EVEN_ODD = 0x1;
static const int ODD_EVEN = 0x2;
static const int ODD_ODD = 0x3;

__host__ __device__ inline int isXodd(int config)
{
	return ((0xFFFFFFFD | config) == 0xFFFFFFFF) ? 1 : 0;
}

__host__ __device__ inline int isYodd(int config)
{
	return ((0xFFFFFFFE | config) == 0xFFFFFFFF) ? 1 : 0;
}

extern "C" __global__ void device_add(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	// implementation in DeviceWrapper.ptx	
}

extern "C" __global__ void device_multiply(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	// implementation in DeviceWrapper.ptx	
}

extern "C" __global__ void device_get_clock(unsigned long long* result)
{
	// implementation in DeviceWrapper.ptx	
}

extern "C" __global__ void device_multiply_partial(unsigned int* result, const unsigned int* x, const unsigned int* y, const int config)
{
	register const int arraySize = BigInteger::ARRAY_SIZE + 1;

	__shared__ unsigned int sharedResult[arraySize];
	__shared__ unsigned int carries[arraySize];		// todo: bank conflict?

	register const int xIndex = threadIdx.x * 2 + isXodd(config);

	sharedResult[xIndex] = 0;
	sharedResult[xIndex + 1] = 0;
	carries[xIndex] = 0;
	carries[xIndex + 1] = 0;

#pragma unroll
	for (register int yIndex = isYodd(config); yIndex < arraySize; yIndex = yIndex + 2)
	{
		if (xIndex + yIndex >= arraySize)
			break;

		register unsigned int carry = carries[xIndex + yIndex];
		carries[xIndex + yIndex] = 0;

		asm volatile (
			"add.cc.u32 %0, %0, %5; \n\t"
			"mad.lo.cc.u32 %0, %3, %4, %0; \n\t"
			"madc.hi.cc.u32 %1, %3, %4, %1; \n\t"
			"addc.u32 %2, %2, 0; \n\t"
			: "+r"(sharedResult[xIndex + yIndex]), "+r"(sharedResult[xIndex + yIndex + 1]), "+r"(carries[xIndex + yIndex + 2])
			: "r"(x[xIndex]), "r"(y[yIndex]), "r"(carry));

		__syncthreads();
	}
			
	result[xIndex] = sharedResult[xIndex];
	result[xIndex + 1] = sharedResult[xIndex + 1];

	__syncthreads();
}

extern "C" __global__ void device_add_partial(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	register const int startIndex = threadIdx.x * DeviceWrapper::ADDITION_CELLS_PER_THREAD;
	__shared__ unsigned short carries[BigInteger::ARRAY_SIZE + 1];	// todo: bank conflict?

	register int index;
#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
	{
		asm volatile (
			"addc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate carry
			: "=r"(result[startIndex + index])
			: "r"(x[startIndex + index]), "r"(y[startIndex + index]));		
	}

	// last iteration generates and stores carry in the array
	asm volatile (
		"addc.cc.u32 %0, %2, %3; \n\t"
		"addc.u16 %1, 0, 0; \n\t"
		: "=r"(result[startIndex + index]), "=h"(carries[startIndex + 1 + index])
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]));

	__syncthreads();	

	register unsigned int carry;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ADDITION_THREAD_COUNT; i++)
	{
		index = 0;
		carry = carries[startIndex + index];

		// first iteration propagates carry from array
		asm volatile (
			"add.cc.u32 %0, %0, %1; \n\t"	//  
			: "+r"(result[startIndex + index])
			: "r"(carry));

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"addc.cc.u32 %0, %0, 0; \n\t"	//propagate generated carries
				: "+r"(result[startIndex + index]));
		}

		// last iteration generates and stores carry in the array
		asm volatile (
			"addc.cc.u32 %0, %0, 0; \n\t"
			"addc.u16 %1, 0, 0; \n\t"
			: "+r"(result[startIndex + index]), "=h"(carries[startIndex + 1 + index]));

		__syncthreads();
	}	
}

inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	}
	return result;
}

DeviceWrapper::DeviceWrapper()
{	
}

DeviceWrapper::~DeviceWrapper()
{
}

unsigned long long DeviceWrapper::getClock(void)
{
	unsigned long long clock;
	unsigned long long* deviceClock;
	checkCuda(hipMalloc(&deviceClock, sizeof(unsigned long long)));
	
	device_get_clock << <1, 1 >> > (deviceClock);

	checkCuda(hipMemcpy(&clock, deviceClock, sizeof(unsigned long long), hipMemcpyDeviceToHost));
	checkCuda(hipFree(deviceClock));
	
	return clock;
}

BigInteger* DeviceWrapper::add(BigInteger& x, BigInteger& y)
{
	//todo: vaildate x,y

	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE + 1];	// + 1 to check for overflow

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;
	
	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size + sizeof(unsigned int)));
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));
	
	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	
	device_add << <1, 1 >> > (device_result, device_x, device_y);
	
	checkCuda(hipMemcpy(resultArray, device_result, size + sizeof(unsigned int), hipMemcpyDeviceToHost));

	unsigned int overflow = resultArray[128];
	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::add overflow!" << endl;
		throw std::overflow_error("BigInteger::add overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));
	
	return new BigInteger(resultArray);;
}

BigInteger* DeviceWrapper::addParallel(BigInteger& x, BigInteger& y)
{
	//todo: vaildate x,y

	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE + 1];	// + 1 to check for overflow

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size + sizeof(unsigned int)));
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT >> > (device_result, device_x, device_y);

	checkCuda(hipMemcpy(resultArray, device_result, size + sizeof(unsigned int), hipMemcpyDeviceToHost));

	unsigned int overflow = resultArray[128];
	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::add overflow!" << endl;
		throw std::overflow_error("BigInteger::add overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	return new BigInteger(resultArray);;
}

BigInteger* DeviceWrapper::multiply(BigInteger& x, BigInteger& y)
{
	//todo: vaildate x,y

	// resultArray twice as long to account for overflow 
	int resultArraySize = BigInteger::ARRAY_SIZE * 2;
	unsigned int* resultArray = new unsigned int[resultArraySize];
	
	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size * 2));	// resultArray twice as long to account for overflow 
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));
		
	device_multiply << <1, 1 >> > (device_result, device_x, device_y);

	checkCuda(hipMemcpy(resultArray, device_result, size * 2, hipMemcpyDeviceToHost));

	unsigned int overflow = 0UL;
	for (int i = resultArraySize - 1; i >= BigInteger::ARRAY_SIZE; i--)
	{
		overflow = overflow | resultArray[i];
	}

	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		throw std::overflow_error("BigInteger::multiply overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	return new BigInteger(resultArray);;
}

BigInteger* DeviceWrapper::multiplyParallel(BigInteger& x, BigInteger& y)
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE + 1]; // + 1 to check for overflow
	
	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;
	int resultArraySize = sizeof(unsigned int) * (BigInteger::ARRAY_SIZE + 1);

	unsigned int* device_result_even_even;
	unsigned int* device_result_even_odd;
	unsigned int* device_result_odd_even;
	unsigned int* device_result_odd_odd;

	unsigned int* device_result_even;
	unsigned int* device_result_odd;

	unsigned int* device_result;

	unsigned int* device_x;
	unsigned int* device_y;
	
	// device memory allocations
	checkCuda(hipMalloc(&device_result_even_even, resultArraySize));
	checkCuda(hipMalloc(&device_result_even_odd, resultArraySize));
	checkCuda(hipMalloc(&device_result_odd_even, resultArraySize));
	checkCuda(hipMalloc(&device_result_odd_odd, resultArraySize));

	checkCuda(hipMalloc(&device_result_even, resultArraySize));
	checkCuda(hipMalloc(&device_result_odd, resultArraySize));
	
	checkCuda(hipMalloc(&device_result, resultArraySize));

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	// two parallel streams for indepentent computations
	hipStream_t evenStream, oddStream;
	checkCuda(hipStreamCreate(&evenStream));
	checkCuda(hipStreamCreate(&oddStream));

	// kernel launches
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, evenStream >> > (device_result_even_even, device_x, device_y, EVEN_EVEN);
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, evenStream >> > (device_result_even_odd, device_x, device_y, EVEN_ODD);
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, oddStream >> > (device_result_odd_even, device_x, device_y, ODD_EVEN);
	device_multiply_partial << <1, DeviceWrapper::MULTIPLICATION_THREAD_COUNT, 0, oddStream >> > (device_result_odd_odd, device_x, device_y, ODD_ODD);

	// reduction
	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT, 0, evenStream >> > (device_result_even, device_result_even_even, device_result_even_odd);
	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT, 0, oddStream >> > (device_result_odd, device_result_odd_even, device_result_odd_odd);

	checkCuda(hipStreamSynchronize(evenStream));
	checkCuda(hipStreamSynchronize(oddStream));

	// reduction
	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT >> > (device_result, device_result_even, device_result_odd);
	
	// copy result to the host
	checkCuda(hipMemcpy(resultArray, device_result, resultArraySize, hipMemcpyDeviceToHost));

	// kill streams
	checkCuda(hipStreamDestroy(evenStream));
	checkCuda(hipStreamDestroy(oddStream));
	
	unsigned int overflow = resultArray[128];
	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		//throw std::overflow_error("BigInteger::multiply overflow");
	}

	// clear memory
	checkCuda(hipFree(device_result_even_even));
	checkCuda(hipFree(device_result_even_odd));
	checkCuda(hipFree(device_result_odd_even));
	checkCuda(hipFree(device_result_odd_odd));

	checkCuda(hipFree(device_result_even));
	checkCuda(hipFree(device_result_odd));

	checkCuda(hipFree(device_result));

	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	BigInteger* result = new BigInteger(resultArray);
	if (result->getLength() < x.getLength() || result->getLength() < y.getLength())
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		//throw std::overflow_error("BigInteger::multiply overflow");
	}

	return result;
}


