#include "hip/hip_runtime.h"
#include "DeviceWrapper.h"
#include "BigInteger.h"
#include "BuildConfig.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

// shared memory stuctures
typedef struct
{
	unsigned int value;	
	unsigned int padding[31];
	// padding to match with 32 byte memory line

} memory32byte;

typedef struct
{
	memory32byte result[DeviceWrapper::ADDITION_CELLS_PER_THREAD];
	unsigned int carry;	
	// 4 byte carry offsets to another memory bank, which eliminates bank conflicts

} additionSharedMemory;

typedef struct
{
	memory32byte result[DeviceWrapper::ADDITION_CELLS_PER_THREAD];
	unsigned int borrow;
	// 4 byte borrow offsets to another memory bank, which eliminates bank conflicts

} subtractionSharedMemory;

//Mapping to sepcific indices of shared memory in order to eliminate bank conflicts in device_multiply_partial
//Dependency: 
// return index % 64 * 32 + (index % 64 & 0xfffffffe) / 2 + index / 64 * 64;
__constant__ unsigned int deviceIndexFixupTable[] { 0, 32, 65, 97, 130, 162, 195, 227, 260, 292, 325, 357,390, 422, 455,
487, 520, 552, 585, 617, 650, 682, 715, 747, 780, 812, 845, 877, 910, 942, 975, 1007, 1040, 1072, 1105, 1137,1170, 1202,
1235, 1267, 1300, 1332, 1365, 1397, 1430, 1462, 1495, 1527, 1560, 1592, 1625, 1657, 1690, 1722, 1755, 1787,1820, 1852,
1885, 1917, 1950, 1982, 2015, 2047, 64, 96, 129, 161, 194, 226, 259, 291, 324, 356, 389, 421, 454, 486, 519,551, 584,
616, 649, 681, 714, 746, 779, 811, 844, 876, 909, 941, 974, 1006, 1039, 1071, 1104, 1136, 1169, 1201, 1234, 1266,1299,
1331, 1364, 1396, 1429, 1461, 1494, 1526, 1559, 1591, 1624, 1656, 1689, 1721, 1754, 1786, 1819, 1851, 1884, 1916, 1949,
1981, 2014, 2046, 2079, 2111, 128 };

__host__ __device__ inline int isXodd(int config)
{
	return ((0xFFFFFFFD | config) == 0xFFFFFFFF) ? 1 : 0;
}

__host__ __device__ inline int isYodd(int config)
{
	return ((0xFFFFFFFE | config) == 0xFFFFFFFF) ? 1 : 0;
}

extern "C" __global__ void device_get_clock(unsigned int* result)
{
	// todo	
}

__host__ __device__ inline bool inBounds128(int index)
{
	return index >= 0 && index <= 127;	
}

extern "C" __global__ void device_clone_partial(unsigned int* x, unsigned int* y)
{
	register int index = threadIdx.x;
	x[index] = y[index];
}

extern "C" __global__ void device_clear_partial(unsigned int* x)
{
	register int index = threadIdx.x;
	x[index] = x[index] ^ x[index];
}

extern "C" __global__ void device_shift_left_partial(unsigned int* x, int n)
{
	register int index = threadIdx.x;
	register int ints = n >> 5;		// n / 32
	register int bits = n & 0x1f;	// n mod 32

	__shared__ unsigned int sharedX[BigInteger::ARRAY_SIZE];
	__shared__ unsigned int sharedResult[BigInteger::ARRAY_SIZE + 1];

	sharedX[index] = inBounds128(index - ints) ? x[index - ints] : 0;
	sharedResult[index] = 0UL;

	__syncthreads();

	register int remainingBits = 32 - bits;
	sharedResult[index + 1] = sharedX[index] >> remainingBits;
	__syncthreads();
	sharedResult[index] = sharedResult[index] | sharedX[index] << bits;
	__syncthreads();

	if (bits > 0)
		x[index] = sharedResult[index];
	else
		x[index] = sharedX[index];	// dummy store - constant time execution
	
	__syncthreads();
}

extern "C" __global__ void device_shift_right_partial(unsigned int* x, int n)
{
	register int index = threadIdx.x;
	register int ints = n >> 5;		// n / 32
	register int bits = n & 0x1f;	// n mod 32

	__shared__ unsigned int sharedX[BigInteger::ARRAY_SIZE + 1];
	__shared__ unsigned int sharedResult[BigInteger::ARRAY_SIZE + 1];

	sharedX[index] = inBounds128(index + ints) ? x[index + ints] : 0UL;	
	sharedResult[index] = 0UL;

	__syncthreads();

	register int remainingBits = 32 - bits;
	if (index - 1 >= 0)
		sharedResult[index - 1] = sharedX[index] << remainingBits;
	else
		sharedResult[127] = 0UL;
	__syncthreads();
	sharedResult[index] = sharedResult[index] | (sharedX[index] >> bits);
	__syncthreads();

	if (bits > 0)
		x[index] = sharedResult[index];
	else
		x[index] = sharedX[index];	// dummy store - constant time execution
	
	__syncthreads();
}

// x and y must 128 unsigned ints allocated
// result return in x
extern "C" __global__ void device_add_partial(unsigned int* x, unsigned int* y)
{
	// offsets to next 'row' of flatten array
	x = x + (blockIdx.x << 8);
	y = y + (blockIdx.x << 8);	

	register const int resultIndex = threadIdx.x;
	register const int startIndex = resultIndex << 2;	// * DeviceWrapper::ADDITION_CELLS_PER_THREAD;

	// 32 threads + 1 to avoid out of bounds exception
	__shared__ additionSharedMemory shared[33];

	register int index = 0;

	asm volatile (
		"add.cc.u32 %0, %1, %2; \n\t"	// first iteration - only genarate carry
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	asm volatile (
		"addc.cc.u32 %0, %1, %2; \n\t"	// propagate and genarate carry
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	asm volatile (
		"addc.cc.u32 %0, %1, %2; \n\t"	// propagate and genarate carry
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	// last iteration generates and stores carry in the array
	asm volatile (
		"addc.cc.u32 %0, %2, %3; \n\t"
		"addc.u32 %1, 0, 0; \n\t"
		: "=r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	__syncthreads();

	register unsigned int carry;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ONE_WARP; i++)
	{
		index = 0;
		carry = shared[resultIndex].carry;

		// first iteration propagates carry from array
		asm volatile (
			"add.cc.u32 %0, %0, %1; \n\t"	//  
			: "+r"(shared[resultIndex].result[index].value)
			: "r"(carry) : "memory");

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"addc.cc.u32 %0, %0, 0; \n\t"	//propagate generated carries
				: "+r"(shared[resultIndex].result[index].value) :: "memory");
		}

		// last iteration generates and stores carry in the array
		asm volatile (
			"addc.cc.u32 %0, %0, 0; \n\t"
			"addc.u32 %1, 0, 0; \n\t"
			: "+r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry) :: "memory");

		__syncthreads();
	}

#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD; index++)
	{
		// store result in x
		x[startIndex + index] = shared[resultIndex].result[index].value;
	}

	__syncthreads();
}

// x and y must 128 unsigned ints allocated
// result return in x
extern "C" __global__ void device_subtract_partial(unsigned int* x, unsigned int* y)
{
	// offsets to next 'row' of flatten array
	x = x + (blockIdx.x << 8);	// * 128 * 2
	y = y + (blockIdx.x << 8);

	register const int resultIndex = threadIdx.x;
	register const int startIndex = resultIndex << 2; // * DeviceWrapper::ADDITION_CELLS_PER_THREAD;

	// 32 threads + 1 to avoid out of bounds exception
	__shared__ subtractionSharedMemory shared[33];

	shared[resultIndex].borrow = 0UL;
	__syncthreads();

	register int index = 0;

	asm volatile (
		"sub.cc.u32 %0, %1, %2; \n\t"	//first interation - only genarate borrow out
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;
		
	asm volatile (
		"subc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate borrow out
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	asm volatile (
		"subc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate borrow out
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");
		
	index++;

	// last iteration generates and stores borrow in the array
	asm volatile (
		"subc.cc.u32 %0, %2, %3; \n\t"
		"subc.u32 %1, 1, 0; \n\t"	// if borrow out than %1 has 0 (1-0-1=0), else %1 has 1 (1-0-0=1)
		"xor.b32 %1, %1, 1; \n\t"	// invert 1-->0 and 0-->1
		: "=r"(shared[resultIndex].result[index].value), "+r"(shared[resultIndex + 1].borrow)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	__syncthreads();

	register unsigned int borrow;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ONE_WARP; i++)
	{
		index = 0;
		borrow = shared[resultIndex].borrow;

		// first iteration propagates borrow from array
		asm volatile (
			"sub.cc.u32 %0, %0, %1; \n\t"
			: "+r"(shared[resultIndex].result[index].value)
			: "r"(borrow) : "memory");

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"subc.cc.u32 %0, %0, 0; \n\t"	//propagate generated borrows
				: "+r"(shared[resultIndex].result[index].value) :: "memory");
		}

		__syncthreads();

		// last iteration generates and stores borrow in the array
		asm volatile (
			"subc.cc.u32 %0, %0, 0; \n\t"
			"subc.u32 %1, 1, 0; \n\t"
			"xor.b32 %1, %1, 1; \n\t"	// invert 1-->0 and 0-->1
			: "+r"(shared[resultIndex].result[index].value), "+r"(shared[resultIndex + 1].borrow) :: "memory");

		__syncthreads();
	}

	
#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD; index++)
	{
		// store result in x
		x[startIndex + index] = shared[resultIndex].result[index].value;
	}

	__syncthreads();
}

extern "C" __global__ void device_multiply_partial(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	register const int arraySize = BigInteger::ARRAY_SIZE;

	// memory banks(32) * (threads(64) + padding(2)) = 32 * 66 = 2112
	__shared__ unsigned int sharedResult[2112];
	__shared__ unsigned int carries[2112];

	// offesets to proper result array index
	result = result + (blockIdx.x << 7); // * arraySize;

	register const int xIndex = (threadIdx.x << 1) + isXodd(blockIdx.x);

	sharedResult[deviceIndexFixupTable[xIndex]] = 0;
	sharedResult[deviceIndexFixupTable[xIndex + 1]] = 0;
	carries[deviceIndexFixupTable[xIndex]] = 0;
	carries[deviceIndexFixupTable[xIndex + 1]] = 0;

#pragma unroll
	for (register int yIndex = isYodd(blockIdx.x); yIndex < arraySize; yIndex = yIndex + 2)
	{
		if (xIndex + yIndex >= arraySize)
			break;

		register unsigned int carry = carries[deviceIndexFixupTable[xIndex + yIndex]];
		carries[deviceIndexFixupTable[xIndex + yIndex]] = 0;

		asm volatile (
			"add.cc.u32 %0, %0, %5; \n\t"
			"mad.lo.cc.u32 %0, %3, %4, %0; \n\t"
			"madc.hi.cc.u32 %1, %3, %4, %1; \n\t"
			"addc.u32 %2, %2, 0; \n\t"
			: "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex]]), "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex + 1]]), "+r"(carries[deviceIndexFixupTable[xIndex + yIndex + 2]])
			: "r"(x[xIndex]), "r"(y[yIndex]), "r"(carry) : "memory");

		__syncthreads();
	}
			 
	result[xIndex] = sharedResult[deviceIndexFixupTable[xIndex]];
	if (xIndex + 1 < 128)	
		result[xIndex + 1] = sharedResult[deviceIndexFixupTable[xIndex + 1]];

	__syncthreads();
}


inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	}
	return result;
}

DeviceWrapper::DeviceWrapper()
{
	checkCuda(hipStreamCreate(&mainStream));
	checkCuda(hipStreamCreate(&memoryCopyStream));
}

DeviceWrapper::~DeviceWrapper()
{
	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipStreamDestroy(mainStream));

	checkCuda(hipStreamSynchronize(memoryCopyStream));
	checkCuda(hipStreamDestroy(memoryCopyStream));
}

unsigned long long DeviceWrapper::getClock(void)
{
	unsigned long long clock;
	unsigned long long* deviceClock;
	checkCuda(hipMalloc(&deviceClock, sizeof(unsigned long long)));
	
//	device_get_clock << <1, 1>> > (deviceClock);

	checkCuda(hipMemcpy(&clock, deviceClock, sizeof(unsigned long long), hipMemcpyDeviceToHost));
	checkCuda(hipFree(deviceClock));
	
	return clock;
}

void DeviceWrapper::clear(BigInteger& x) const
{
	int size = sizeof(unsigned int) << 7;	// * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMemcpyAsync(device_x, x.magnitude, size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(BigInteger::ARRAY_SIZE);	// 128

	device_clear_partial << <blocks, threads, 0, mainStream >> > (device_x);

	checkCuda(hipMemcpyAsync(x.magnitude, device_x, size, hipMemcpyDeviceToHost, mainStream));

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_x));
}

// x := y
void DeviceWrapper::clone(BigInteger& x, const BigInteger& y) const
{
	int size = sizeof(unsigned int) << 7; // * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpyAsync(device_x, x.magnitude, size, hipMemcpyHostToDevice, memoryCopyStream));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpyAsync(device_y, y.magnitude, size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(BigInteger::ARRAY_SIZE);	//128

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_clone_partial << <blocks, threads, 0, mainStream >> > (device_x, device_y);

	checkCuda(hipEventDestroy(event));

	checkCuda(hipMemcpyAsync(x.magnitude, device_x, size, hipMemcpyDeviceToHost, mainStream));
	checkCuda(hipFree(device_y));

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_x));
}

void DeviceWrapper::shiftLeftParallel(BigInteger& x, int bits) const
{	
	int size = sizeof(unsigned int) << 7;	// * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMemcpyAsync(device_x, x.magnitude, size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(BigInteger::ARRAY_SIZE);	// 128

	device_shift_left_partial << <blocks, threads, 0, mainStream >> > (device_x, bits);

	checkCuda(hipMemcpyAsync(x.magnitude, device_x, size, hipMemcpyDeviceToHost, mainStream));

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_x));
}

void DeviceWrapper::shiftRightParallel(BigInteger& x, int bits) const
{	
	int size = sizeof(unsigned int) << 7;	// * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMemcpyAsync(device_x, x.magnitude, size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(BigInteger::ARRAY_SIZE);	// 128

	device_shift_right_partial << <blocks, threads, 0, mainStream >> > (device_x, bits);

	checkCuda(hipMemcpyAsync(x.magnitude, device_x, size, hipMemcpyDeviceToHost, mainStream));

	checkCuda(hipStreamSynchronize(mainStream));	
	checkCuda(hipFree(device_x));
}

void DeviceWrapper::addParallel(BigInteger& x, const BigInteger& y) const
{	
	int size = sizeof(unsigned int) << 7;	// * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpyAsync(device_x, x.magnitude, size, hipMemcpyHostToDevice, memoryCopyStream));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpyAsync(device_y, y.magnitude, size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(DeviceWrapper::ONE_WARP);

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_add_partial << <blocks, threads, 0, mainStream >> > (device_x, device_y);

	checkCuda(hipEventDestroy(event));

	if (DEBUG)
	{
		unsigned int* resultArray = new unsigned int[128];
		checkCuda(hipMemcpyAsync(resultArray, device_x, size, hipMemcpyDeviceToHost, mainStream));
		checkCuda(hipFree(device_y));

		checkCuda(hipStreamSynchronize(mainStream));
		checkCuda(hipFree(device_x));

		// analizing result's length with inputs' lengths
		// to detect possible overflow
		int resultLength = 128, xLength = 128, yLength = 128;
		bool resultSet = false, xSet = false, ySet = false;
		for (int i = 127; i >= 0; i--)
		{
			if (x.magnitude[i] == 0UL && !xSet)
				xLength--;
			else
				xSet = true;

			if (y.magnitude[i] == 0UL && !ySet)
				yLength--;
			else
				ySet = true;

			if (resultArray[i] == 0UL && !resultSet)
				resultLength--;
			else
				resultSet = true;

			x.magnitude[i] = resultArray[i];
		}

		if (resultLength < xLength || resultLength < yLength)
		{
			std::cerr << "ERROR: BigInteger::add overflow! -- length difference" << endl;
		}
	}
	else
	{		
		checkCuda(hipMemcpyAsync(x.magnitude, device_x, size, hipMemcpyDeviceToHost, mainStream));
		checkCuda(hipFree(device_y));

		checkCuda(hipStreamSynchronize(mainStream));
		checkCuda(hipFree(device_x));
	}
}

void DeviceWrapper::subtractParallel(BigInteger& x, const BigInteger& y) const
{
	if (DEBUG)
	{
		if (x.compare(y) != -1) // if y isn't lower
		{
			std::cerr << "ERROR: BigInteger::subtract - negitve output" << endl;
		}
	}

	int size = sizeof(unsigned int) << 7; // * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpyAsync(device_x, x.magnitude, size, hipMemcpyHostToDevice, memoryCopyStream));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpyAsync(device_y, y.magnitude, size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(DeviceWrapper::ONE_WARP);

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_subtract_partial << <blocks, threads, 0, mainStream >> > (device_x, device_y);

	checkCuda(hipEventDestroy(event));

	checkCuda(hipMemcpyAsync(x.magnitude, device_x, size, hipMemcpyDeviceToHost, mainStream));
	checkCuda(hipFree(device_y));

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_x));	
}

void DeviceWrapper::multiplyParallel(BigInteger& x, const BigInteger& y) const
{
	int size = sizeof(unsigned int) << 7; // * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	// device memory allocations
	checkCuda(hipMalloc(&device_result, (size << 2)));	// 4 times for every block
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpy(device_x, x.magnitude, size, hipMemcpyHostToDevice));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpy(device_y, y.magnitude, size, hipMemcpyHostToDevice));

	// launch config
	dim3 blocks(DeviceWrapper::MULTIPLICATION_BLOCKS_COUNT);
	dim3 threads(DeviceWrapper::TWO_WARPS);

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_multiply_partial << <blocks, threads, 0, mainStream>> > (device_result, device_x, device_y);
	
	checkCuda(hipEventDestroy(event));

	// reduction
	blocks.x = 2;
	threads.x = DeviceWrapper::ONE_WARP;
	device_add_partial << <blocks, threads, 0, mainStream >> > (device_result, device_result + 128);

	// reduction
	blocks.x = 1;
	device_add_partial << <blocks, threads, 0, mainStream >> > (device_result, device_result + 256);
		
	if (DEBUG)
	{
		unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];

		checkCuda(hipMemcpyAsync(resultArray, device_result, size, hipMemcpyDeviceToHost, mainStream));
		
		checkCuda(hipFree(device_x));
		checkCuda(hipFree(device_y));

		checkCuda(hipStreamSynchronize(mainStream));
		checkCuda(hipFree(device_result));

		// analizing result's length with inputs' lengths
		// to detect possible overflow
		int resultLength = 128, xLength = 128, yLength = 128;
		bool resultSet = false, xSet = false, ySet = false;
		for (int i = 127; i >= 0; i--)
		{
			if (x.magnitude[i] == 0UL && !xSet)
				xLength--;
			else
				xSet = true;

			if (y.magnitude[i] == 0UL && !ySet)
				yLength--;
			else
				ySet = true;

			if (resultArray[i] == 0UL && !resultSet)
				resultLength--;
			else
				resultSet = true;

			x.magnitude[i] = resultArray[i];
		}

		if (resultLength < xLength || resultLength < yLength)
		{
			std::cerr << "ERROR: BigInteger::multiply overflow! -- length difference" << endl;
		}
	}
	else
	{
		// copy result to the host
		checkCuda(hipMemcpyAsync(x.magnitude, device_result, size, hipMemcpyDeviceToHost, mainStream));

		// clear memory
		checkCuda(hipFree(device_x));
		checkCuda(hipFree(device_y));

		checkCuda(hipStreamSynchronize(mainStream));
		checkCuda(hipFree(device_result));
	}
}


