#include "hip/hip_runtime.h"
#include "DeviceWrapper.h"
#include "BigInteger.h"
#include "BuildConfig.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

// shared memory stuctures
typedef struct
{
	unsigned int value;	
	unsigned int padding[31];
	// padding to match with 32 byte memory line

} memory32byte;

typedef struct
{
	memory32byte result[DeviceWrapper::ADDITION_CELLS_PER_THREAD];
	unsigned int carry;	
	// 4 byte carry offsets to another memory bank, which eliminates bank conflicts

} additionSharedMemory;

typedef struct
{
	memory32byte result[DeviceWrapper::ADDITION_CELLS_PER_THREAD];
	unsigned int borrow;
	// 4 byte borrow offsets to another memory bank, which eliminates bank conflicts

} subtractionSharedMemory;

//Mapping to sepcific indices of shared memory in order to eliminate bank conflicts in device_multiply_partial
//Dependency: 
// return index % 64 * 32 + (index % 64 & 0xfffffffe) / 2 + index / 64 * 64;
__constant__ unsigned int deviceIndexFixupTable[] { 0, 32, 65, 97, 130, 162, 195, 227, 260, 292, 325, 357,390, 422, 455,
487, 520, 552, 585, 617, 650, 682, 715, 747, 780, 812, 845, 877, 910, 942, 975, 1007, 1040, 1072, 1105, 1137,1170, 1202,
1235, 1267, 1300, 1332, 1365, 1397, 1430, 1462, 1495, 1527, 1560, 1592, 1625, 1657, 1690, 1722, 1755, 1787,1820, 1852,
1885, 1917, 1950, 1982, 2015, 2047, 64, 96, 129, 161, 194, 226, 259, 291, 324, 356, 389, 421, 454, 486, 519,551, 584,
616, 649, 681, 714, 746, 779, 811, 844, 876, 909, 941, 974, 1006, 1039, 1071, 1104, 1136, 1169, 1201, 1234, 1266,1299,
1331, 1364, 1396, 1429, 1461, 1494, 1526, 1559, 1591, 1624, 1656, 1689, 1721, 1754, 1786, 1819, 1851, 1884, 1916, 1949,
1981, 2014, 2046, 2079, 2111, 128 };

__host__ __device__ inline int isXodd(int config)
{
	return ((0xFFFFFFFD | config) == 0xFFFFFFFF) ? 1 : 0;
}

__host__ __device__ inline int isYodd(int config)
{
	return ((0xFFFFFFFE | config) == 0xFFFFFFFF) ? 1 : 0;
}

extern "C" __global__ void device_get_clock(unsigned int* result)
{
	// todo	
}

// x and y must 128 unsigned ints allocated
// result return in x
extern "C" __global__ void device_add_partial(unsigned int* x, unsigned int* y)
{
	// offsets to next 'row' of flatten array
	x = x + blockIdx.x * 256;
	y = y + blockIdx.x * 256;

	register const int resultIndex = threadIdx.x;
	register const int startIndex = resultIndex * DeviceWrapper::ADDITION_CELLS_PER_THREAD;

	// 32 threads + 1 to avoid out of bounds exception
	__shared__ additionSharedMemory shared[33];

	register int index = 0;

	asm volatile (
		"add.cc.u32 %0, %1, %2; \n\t"	// first iteration - only genarate carry
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	asm volatile (
		"addc.cc.u32 %0, %1, %2; \n\t"	// propagate and genarate carry
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	asm volatile (
		"addc.cc.u32 %0, %1, %2; \n\t"	// propagate and genarate carry
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	// last iteration generates and stores carry in the array
	asm volatile (
		"addc.cc.u32 %0, %2, %3; \n\t"
		"addc.u32 %1, 0, 0; \n\t"
		: "=r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	__syncthreads();

	register unsigned int carry;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ONE_WARP; i++)
	{
		index = 0;
		carry = shared[resultIndex].carry;

		// first iteration propagates carry from array
		asm volatile (
			"add.cc.u32 %0, %0, %1; \n\t"	//  
			: "+r"(shared[resultIndex].result[index].value)
			: "r"(carry) : "memory");

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"addc.cc.u32 %0, %0, 0; \n\t"	//propagate generated carries
				: "+r"(shared[resultIndex].result[index].value) :: "memory");
		}

		// last iteration generates and stores carry in the array
		asm volatile (
			"addc.cc.u32 %0, %0, 0; \n\t"
			"addc.u32 %1, 0, 0; \n\t"
			: "+r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry) :: "memory");

		__syncthreads();
	}

#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD; index++)
	{
		// store result in x
		x[startIndex + index] = shared[resultIndex].result[index].value;
	}

	__syncthreads();
}

// x and y must 128 unsigned ints allocated
// result return in x
extern "C" __global__ void device_subtract_partial(unsigned int* x, unsigned int* y)
{
	// offsets to next 'row' of flatten array
	x = x + blockIdx.x * 256;
	y = y + blockIdx.x * 256;

	register const int resultIndex = threadIdx.x;
	register const int startIndex = resultIndex * DeviceWrapper::ADDITION_CELLS_PER_THREAD;

	// 32 threads + 1 to avoid out of bounds exception
	__shared__ subtractionSharedMemory shared[33];

	register int index = 0;

	asm volatile (
		"sub.cc.u32 %0, %1, %2; \n\t"	//first interation - only genarate borrow out
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;
		
	asm volatile (
		"subc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate borrow out
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	index++;

	asm volatile (
		"subc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate borrow out
		: "=r"(shared[resultIndex].result[index].value)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");
		
	index++;

	// last iteration generates and stores borrow in the array
	asm volatile (
		"subc.cc.u32 %0, %2, %3; \n\t"
		"subc.u32 %1, 1, 0; \n\t"	// if borrow out than %1 has 0 (1-0-1=0), else %1 has 1 (1-0-0=1)
		"xor.b32 %1, %1, 1; \n\t"	// invert 1-->0 and 0-->1
		: "=r"(shared[resultIndex].result[index].value), "+r"(shared[resultIndex + 1].borrow)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]) : "memory");

	__syncthreads();

	register unsigned int borrow;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ONE_WARP; i++)
	{
		index = 0;
		borrow = shared[resultIndex].borrow;

		// first iteration propagates borrow from array
		asm volatile (
			"sub.cc.u32 %0, %0, %1; \n\t"
			: "+r"(shared[resultIndex].result[index].value)
			: "r"(borrow) : "memory");

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"subc.cc.u32 %0, %0, 0; \n\t"	//propagate generated borrows
				: "+r"(shared[resultIndex].result[index].value) :: "memory");
		}

		__syncthreads();

		// last iteration generates and stores borrow in the array
		asm volatile (
			"subc.cc.u32 %0, %0, 0; \n\t"
			"subc.u32 %1, 1, 0; \n\t"
			"xor.b32 %1, %1, 1; \n\t"	// invert 1-->0 and 0-->1
			: "+r"(shared[resultIndex].result[index].value), "+r"(shared[resultIndex + 1].borrow) :: "memory");

		__syncthreads();
	}

	
#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD; index++)
	{
		// store result in x
		x[startIndex + index] = shared[resultIndex].result[index].value;
	}

	__syncthreads();
}

extern "C" __global__ void device_multiply_partial(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	register const int arraySize = BigInteger::ARRAY_SIZE;

	// memory banks(32) * (threads(64) + padding(2)) = 32 * 66 = 2112
	__shared__ unsigned int sharedResult[2112];
	__shared__ unsigned int carries[2112];

	// offesets to proper result array index
	result = result + blockIdx.x * arraySize;

	register const int xIndex = threadIdx.x * 2 + isXodd(blockIdx.x);

	sharedResult[deviceIndexFixupTable[xIndex]] = 0;
	sharedResult[deviceIndexFixupTable[xIndex + 1]] = 0;
	carries[deviceIndexFixupTable[xIndex]] = 0;
	carries[deviceIndexFixupTable[xIndex + 1]] = 0;

#pragma unroll
	for (register int yIndex = isYodd(blockIdx.x); yIndex < arraySize; yIndex = yIndex + 2)
	{
		if (xIndex + yIndex >= arraySize)
			break;

		register unsigned int carry = carries[deviceIndexFixupTable[xIndex + yIndex]];
		carries[deviceIndexFixupTable[xIndex + yIndex]] = 0;

		asm volatile (
			"add.cc.u32 %0, %0, %5; \n\t"
			"mad.lo.cc.u32 %0, %3, %4, %0; \n\t"
			"madc.hi.cc.u32 %1, %3, %4, %1; \n\t"
			"addc.u32 %2, %2, 0; \n\t"
			: "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex]]), "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex + 1]]), "+r"(carries[deviceIndexFixupTable[xIndex + yIndex + 2]])
			: "r"(x[xIndex]), "r"(y[yIndex]), "r"(carry) : "memory");

		__syncthreads();
	}
			 
	result[xIndex] = sharedResult[deviceIndexFixupTable[xIndex]];
	if (xIndex + 1 < 128)	
		result[xIndex + 1] = sharedResult[deviceIndexFixupTable[xIndex + 1]];

	__syncthreads();
}


inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	}
	return result;
}

DeviceWrapper::DeviceWrapper()
{
	checkCuda(hipStreamCreate(&mainStream));
	checkCuda(hipStreamCreate(&memoryCopyStream));
}

DeviceWrapper::~DeviceWrapper()
{
	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipStreamDestroy(mainStream));

	checkCuda(hipStreamSynchronize(memoryCopyStream));
	checkCuda(hipStreamDestroy(memoryCopyStream));
}

unsigned long long DeviceWrapper::getClock(void)
{
	unsigned long long clock;
	unsigned long long* deviceClock;
	checkCuda(hipMalloc(&deviceClock, sizeof(unsigned long long)));
	
//	device_get_clock << <1, 1>> > (deviceClock);

	checkCuda(hipMemcpy(&clock, deviceClock, sizeof(unsigned long long), hipMemcpyDeviceToHost));
	checkCuda(hipFree(deviceClock));
	
	return clock;
}

unsigned int* DeviceWrapper::addParallel(const BigInteger& x, const BigInteger& y) const
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];	

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;	

	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpyAsync(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice, memoryCopyStream));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpyAsync(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(DeviceWrapper::ONE_WARP);

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_add_partial << <blocks, threads, 0, mainStream >> > (device_x, device_y);

	checkCuda(hipEventDestroy(event));

	checkCuda(hipMemcpyAsync(resultArray, device_x, size, hipMemcpyDeviceToHost, mainStream));	
	checkCuda(hipFree(device_y));	

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_x));	

	if (DEBUG)
	{
		// analizing result's length with inputs' lengths
		// to detect possible overflow
		int resultLength = 128, xLength = 128, yLength = 128;
		bool resultSet = false, xSet = false, ySet = false;
		for (int i = 127; i >= 0; i--)
		{
			if (x.getMagnitudeArray()[i] == 0UL && !xSet)
				xLength--;
			else
				xSet = true;

			if (y.getMagnitudeArray()[i] == 0UL && !ySet)
				yLength--;
			else
				ySet = true;

			if (resultArray[i] == 0UL && !resultSet)
				resultLength--;
			else
				resultSet = true;
		}

		if (resultLength < xLength || resultLength < yLength)
		{
			std::cerr << "ERROR: BigInteger::add overflow! -- length difference" << endl;
		}
	}

	return resultArray;
}

unsigned int* DeviceWrapper::subtractParallel(const BigInteger& x, const BigInteger& y) const
{
	if (DEBUG)
	{
		if (x.compare(y) != -1) // if y isn't lower
		{
			std::cerr << "ERROR: BigInteger::subtract - negitve output" << endl;
		}
	}

	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;

	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpyAsync(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice, memoryCopyStream));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpyAsync(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice, mainStream));

	// launch config
	dim3 blocks(1);
	dim3 threads(DeviceWrapper::ONE_WARP);

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_subtract_partial << <blocks, threads, 0, mainStream >> > (device_x, device_y);

	checkCuda(hipEventDestroy(event));

	checkCuda(hipMemcpyAsync(resultArray, device_x, size, hipMemcpyDeviceToHost, mainStream));
	checkCuda(hipFree(device_y));

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_x));

	return resultArray;
}

unsigned int* DeviceWrapper::multiplyParallel(const BigInteger& x, const BigInteger& y) const
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	// device memory allocations
	checkCuda(hipMalloc(&device_result, size * 4));	// 4 times for every block
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	hipEvent_t event;
	checkCuda(hipEventCreate(&event));

	// async memory copy
	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipEventRecord(event, memoryCopyStream));	// record x copy finish
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	// launch config
	dim3 blocks(DeviceWrapper::MULTIPLICATION_BLOCKS_COUNT);
	dim3 threads(DeviceWrapper::TWO_WARPS);

	checkCuda(hipStreamWaitEvent(mainStream, event, 0));	// wait for x,y to finish
	device_multiply_partial << <blocks, threads, 0, mainStream>> > (device_result, device_x, device_y);
	
	checkCuda(hipEventDestroy(event));

	// reduction
	blocks.x = 2;
	threads.x = DeviceWrapper::ONE_WARP;
	device_add_partial << <blocks, threads, 0, mainStream >> > (device_result, device_result + 128);

	// reduction
	blocks.x = 1;
	device_add_partial << <blocks, threads, 0, mainStream >> > (device_result, device_result + 256);
	
	// copy result to the host
	checkCuda(hipMemcpyAsync(resultArray, device_result, size, hipMemcpyDeviceToHost, mainStream));
	
	// clear memory
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	checkCuda(hipStreamSynchronize(mainStream));
	checkCuda(hipFree(device_result));	

	if (DEBUG)
	{
		// analizing result's length with inputs' lengths
		// to detect possible overflow
		int resultLength = 128, xLength = 128, yLength = 128;
		bool resultSet = false, xSet = false, ySet = false;
		for (int i = 127; i >= 0; i--)
		{
			if (x.getMagnitudeArray()[i] == 0UL && !xSet)
				xLength--;
			else
				xSet = true;

			if (y.getMagnitudeArray()[i] == 0UL && !ySet)
				yLength--;
			else
				ySet = true;

			if (resultArray[i] == 0UL && !resultSet)
				resultLength--;
			else
				resultSet = true;
		}

		if (resultLength < xLength || resultLength < yLength)
		{
			std::cerr << "ERROR: BigInteger::multiply overflow! -- length difference" << endl;
		}
	}

	return resultArray;
}


