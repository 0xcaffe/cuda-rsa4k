#include "hip/hip_runtime.h"
#include "DeviceWrapper.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void device_add(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	// implementation in DeviceWrapper.ptx	
}

__global__ void device_multiply(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	// implementation in DeviceWrapper.ptx	
}

inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	}
	return result;
}

DeviceWrapper::DeviceWrapper()
{	
}

DeviceWrapper::~DeviceWrapper()
{
}

BigInteger* DeviceWrapper::add(BigInteger& x, BigInteger& y)
{
	//todo: vaildate x,y

	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE + 1];	// + 1 to check for overflow

	int size = sizeof(unsigned int*) * BigInteger::ARRAY_SIZE;
	
	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size + sizeof(unsigned int*)));
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));
	
	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	
	device_add << <1, 1 >> > (device_result, device_x, device_y);
	
	checkCuda(hipMemcpy(resultArray, device_result, size + sizeof(unsigned int*), hipMemcpyDeviceToHost));

	unsigned int overflow = resultArray[128];
	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::add overflow!" << endl;
		throw std::overflow_error("BigInteger::add overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));
	
	return new BigInteger(resultArray);;
}

BigInteger* DeviceWrapper::multiply(BigInteger& x, BigInteger& y)
{
	//todo: vaildate x,y

	// resultArray twice as long to account for overflow 
	int resultArraySize = BigInteger::ARRAY_SIZE * 2;
	unsigned int* resultArray = new unsigned int[resultArraySize];
	
	int size = sizeof(unsigned int*) * BigInteger::ARRAY_SIZE;

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_result, size * 2));	// resultArray twice as long to account for overflow 
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));
		
	device_multiply << <1, 1 >> > (device_result, device_x, device_y);

	checkCuda(hipMemcpy(resultArray, device_result, size * 2, hipMemcpyDeviceToHost));

	unsigned int overflow = 0UL;
	for (int i = resultArraySize - 1; i >= BigInteger::ARRAY_SIZE; i--)
	{
		overflow = overflow | resultArray[i];
	}

	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		throw std::overflow_error("BigInteger::multiply overflow");
	}

	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	return new BigInteger(resultArray);;
}



