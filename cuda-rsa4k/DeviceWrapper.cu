#include "hip/hip_runtime.h"
#include "DeviceWrapper.h"
#include "BigInteger.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

typedef struct
{
	unsigned int value;	
	unsigned int padding[31];
	// padding to match with 32 byte memory line

} memory32byte;

typedef struct
{
	memory32byte result[DeviceWrapper::ADDITION_CELLS_PER_THREAD];
	unsigned int carry;	
	// 4 byte carry offsets to another memory bank, which eliminates bank conflicts

} additionSharedMemory;

__host__ __device__ inline int isXodd(int config)
{
	return ((0xFFFFFFFD | config) == 0xFFFFFFFF) ? 1 : 0;
}

__host__ __device__ inline int isYodd(int config)
{
	return ((0xFFFFFFFE | config) == 0xFFFFFFFF) ? 1 : 0;
}

extern "C" __global__ void device_get_clock(unsigned long long* result)
{
	// todo	
}

// x and y must (128 + 1) unsigned ints allocated to account for overflow
// result return in x
extern "C" __global__ void device_add_partial(unsigned int* x, unsigned int* y)
{
	x = x + blockIdx.x * 2 * (BigInteger::ARRAY_SIZE + 1);
	y = y + blockIdx.x * 2 * (BigInteger::ARRAY_SIZE + 1);

	register const int resultIndex = threadIdx.x;
	register const int startIndex = resultIndex * DeviceWrapper::ADDITION_CELLS_PER_THREAD;

	__shared__ additionSharedMemory shared[BigInteger::ARRAY_SIZE / DeviceWrapper::ADDITION_CELLS_PER_THREAD + 1];

	register int index;

#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
	{
		asm volatile (
			"addc.cc.u32 %0, %1, %2; \n\t"	// genarate and propagate carry
			: "=r"(shared[resultIndex].result[index].value)
			: "r"(x[startIndex + index]), "r"(y[startIndex + index]));
	}

	// last iteration generates and stores carry in the array
	asm volatile (
		"addc.cc.u32 %0, %2, %3; \n\t"
		"addc.u32 %1, 0, 0; \n\t"
		: "=r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry)
		: "r"(x[startIndex + index]), "r"(y[startIndex + index]));

	__syncthreads();

	register unsigned int carry;
#pragma unroll
	for (register int i = 0; i < DeviceWrapper::ADDITION_THREAD_COUNT; i++)
	{
		index = 0;
		carry = shared[resultIndex].carry;

		// first iteration propagates carry from array
		asm volatile (
			"add.cc.u32 %0, %0, %1; \n\t"	//  
			: "+r"(shared[resultIndex].result[index].value)
			: "r"(carry));

#pragma unroll
		for (index = 1; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD - 1; index++)
		{
			asm volatile (
				"addc.cc.u32 %0, %0, 0; \n\t"	//propagate generated carries
				: "+r"(shared[resultIndex].result[index].value));
		}

		// last iteration generates and stores carry in the array
		asm volatile (
			"addc.cc.u32 %0, %0, 0; \n\t"
			"addc.u32 %1, 0, 0; \n\t"
			: "+r"(shared[resultIndex].result[index].value), "=r"(shared[resultIndex + 1].carry));

		__syncthreads();
	}

#pragma unroll
	for (index = 0; index < DeviceWrapper::ADDITION_CELLS_PER_THREAD; index++)
	{
		// store result in x
		x[startIndex + index] = shared[resultIndex].result[index].value;
	}

	__syncthreads();
}



__constant__ unsigned int deviceIndexFixupTable[129];


extern "C" __global__ void device_multiply_partial(unsigned int* result, const unsigned int* x, const unsigned int* y)
{
	register const int arraySize = BigInteger::ARRAY_SIZE + 1;
	register const int sharedMemoryLines = DeviceWrapper::MULTIPLICATION_THREAD_COUNT + 2;
	register const int memoryBanksCount = 32;

	__shared__ unsigned int sharedResult[memoryBanksCount * sharedMemoryLines];
	__shared__ unsigned int carries[memoryBanksCount * sharedMemoryLines];

	// offesets to proper result array index
	result = result + blockIdx.x * (BigInteger::ARRAY_SIZE + 1);	

	register const int xIndex = threadIdx.x * 2 + isXodd(blockIdx.x);

	sharedResult[deviceIndexFixupTable[xIndex]] = 0;
	sharedResult[deviceIndexFixupTable[xIndex + 1]] = 0;
	carries[deviceIndexFixupTable[xIndex]] = 0;
	carries[deviceIndexFixupTable[xIndex + 1]] = 0;

#pragma unroll
	for (register int yIndex = isYodd(blockIdx.x); yIndex < arraySize; yIndex = yIndex + 2)
	{
		if (xIndex + yIndex >= arraySize)
			break;

		register unsigned int carry = carries[deviceIndexFixupTable[xIndex + yIndex]];
		carries[deviceIndexFixupTable[xIndex + yIndex]] = 0;

		asm volatile (
			"add.cc.u32 %0, %0, %5; \n\t"
			"mad.lo.cc.u32 %0, %3, %4, %0; \n\t"
			"madc.hi.cc.u32 %1, %3, %4, %1; \n\t"
			"addc.u32 %2, %2, 0; \n\t"
			: "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex]]), "+r"(sharedResult[deviceIndexFixupTable[xIndex + yIndex + 1]]), "+r"(carries[deviceIndexFixupTable[xIndex + yIndex + 2]])
			: "r"(x[xIndex]), "r"(y[yIndex]), "r"(carry));

		__syncthreads();
	}
			
	result[xIndex] = sharedResult[deviceIndexFixupTable[xIndex]];
	result[xIndex + 1] = sharedResult[deviceIndexFixupTable[xIndex + 1]];

	__syncthreads();
}


inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	}
	return result;
}

DeviceWrapper::DeviceWrapper()
{
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(deviceIndexFixupTable), indexFixupTable, sizeof(unsigned int) * 129));
}

DeviceWrapper::~DeviceWrapper()
{
	delete[] indexFixupTable;
}

unsigned long long DeviceWrapper::getClock(void)
{
	unsigned long long clock;
	unsigned long long* deviceClock;
	checkCuda(hipMalloc(&deviceClock, sizeof(unsigned long long)));
	
	device_get_clock << <1, 1 >> > (deviceClock);

	checkCuda(hipMemcpy(&clock, deviceClock, sizeof(unsigned long long), hipMemcpyDeviceToHost));
	checkCuda(hipFree(deviceClock));
	
	return clock;
}

unsigned int* DeviceWrapper::addParallel(const BigInteger& x, const BigInteger& y)
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];	

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;	

	unsigned int* device_x;
	unsigned int* device_y;

	checkCuda(hipMalloc(&device_x, size + sizeof(unsigned int)));	// + 1 to check for overflow
	checkCuda(hipMalloc(&device_y, size + sizeof(unsigned int)));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	device_add_partial << <1, DeviceWrapper::ADDITION_THREAD_COUNT >> > (device_x, device_y);

	checkCuda(hipMemcpy(resultArray, device_x, size, hipMemcpyDeviceToHost));

	unsigned int overflow;
	checkCuda(hipMemcpy(&overflow, device_x + BigInteger::ARRAY_SIZE, sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::add overflow!" << endl;
		throw std::overflow_error("BigInteger::add overflow");
	}

	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	return resultArray;
}

unsigned int* DeviceWrapper::multiplyParallel(const BigInteger& x, const BigInteger& y)
{
	unsigned int* resultArray = new unsigned int[BigInteger::ARRAY_SIZE];

	int size = sizeof(unsigned int) * BigInteger::ARRAY_SIZE;
	int deviceResultArraySize = size + sizeof(unsigned int);

	unsigned int* device_result;
	unsigned int* device_x;
	unsigned int* device_y;

	// device memory allocations
	checkCuda(hipMalloc(&device_result, deviceResultArraySize * 4));	// 4 times for every block
	checkCuda(hipMalloc(&device_x, size));
	checkCuda(hipMalloc(&device_y, size));

	checkCuda(hipMemcpy(device_x, x.getMagnitudeArray(), size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_y, y.getMagnitudeArray(), size, hipMemcpyHostToDevice));

	dim3 blocks(DeviceWrapper::MULTIPLICATION_BLOCKS_COUNT);
	dim3 threads(DeviceWrapper::MULTIPLICATION_THREAD_COUNT);

	device_multiply_partial << <blocks, threads>> > (device_result, device_x, device_y);

	// reduction
	blocks.x = 2;
	threads.x = DeviceWrapper::ADDITION_THREAD_COUNT;
	device_add_partial << <blocks, threads >> > (device_result, device_result + 129);

	// reduction
	blocks.x = 1;
	device_add_partial << <blocks, threads >> > (device_result, device_result + 258);
	
	// copy result to the host
	checkCuda(hipMemcpy(resultArray, device_result, size, hipMemcpyDeviceToHost));
	
	unsigned int overflow;
	checkCuda(hipMemcpy(&overflow, device_result + BigInteger::ARRAY_SIZE, sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (overflow != 0UL)
	{
		std::cerr << "ERROR: BigInteger::multiply overflow!" << endl;
		//throw std::overflow_error("BigInteger::multiply overflow");
	}

	// clear memory
	checkCuda(hipFree(device_result));
	checkCuda(hipFree(device_x));
	checkCuda(hipFree(device_y));

	// todo overflow?

	return resultArray;
}


